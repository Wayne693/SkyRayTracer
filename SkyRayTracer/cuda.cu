#include "hip/hip_runtime.h"
#include "cuda.cuh"
#include "utility.h"
#include "aarect.h"
#include "material.h"
#include "Box.h"
#include "Sphere.h"
#include "hittable.h"
#include "Camera.h"
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include "thrust/extrema.h"

__constant__ float INF;
__constant__ float PI;

__device__ color raycolor(Ray& ray, const color& backGround, const HittableList** scene, Hittable* light)
{
	//color finalcolor = backGround;
	color n = vec3(0, 0, 0);
	color m = vec3(1, 1, 1);

	//Ray r = ray;


	for (int i = 0; i < 100; i++)
	{
		HitRecord rec;
		scatter_record srec;
		color curcolor;
		if (!(*scene)->hit(ray, 0.01f, INF, rec))
		{
			return n + m * backGround;
		}

		color emitted = rec.mat_ptr->emitted(ray, rec, rec.u, rec.v, rec.p);
		
		if (!rec.mat_ptr->scatter(ray, rec, srec))
		{
			return n + m * emitted;
		}

		pdf* light_pdf = &HittablePDF(light, rec.p);
		pdf* cos_pdf = &CosPDF(rec.normal);
		MixPDF p(light_pdf, cos_pdf);
		
		auto scattered = Ray(rec.p, p.generate(ray.randstate()), ray.randstate());

		auto pdfval = p.value(scattered.direction());
		auto ndotwi = dot(rec.normal.normalized(), scattered.direction().normalized());
		
		n = n + m * emitted;
		m = m * rec.mat_ptr->scattering_pdf(ray, rec, scattered) * thrust::max(0.f, ndotwi) / pdfval;
		ray = scattered;
	}

	return color(0, 0, 0);
}

__global__ void render(vec3* fb, hiprandState* cudaRandState, HittableList** scene, Camera** camera, int width, int height, int spp)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
	{
		return;
	}

	int pixel_index = y * width + x;
	

	//hiprandState currentrs;
	

	hiprand_init(2023, pixel_index, 0, &cudaRandState[pixel_index]);
	//printf("%lf %lf %lf\n", (*camera)->origin.x(), (*camera)->origin.y(), (*camera)->origin.z());
	//printf("%d\n", (*scene)->length);
	
	//printf("%lf %lf %lf %lf %lf\n", r.direction().x(), r.direction().y(), r.direction().z(), u, v);
	auto lm = new DiffuseLight(color(15, 15, 15));
	auto light = RectXZ(213, 343, 227, 332, 554, lm);
	
	color sumcolor = vec3(0, 0, 0);
	for (int i = 0; i < spp; i++)
	{
		float offset_x = Random(cudaRandState[pixel_index]);
		float offset_y = Random(cudaRandState[pixel_index]);
		//printf("%d %lf %lf\n", pixel_index, offset_x, offset_y);
		float u = float(x + offset_x) / float(width);
		float v = float(y + offset_y) / float(height);
		Ray r = (*camera)->GetRay(u, v, cudaRandState[pixel_index]);
		//if (pixel_index == 10584)
		//	printf("idx = %d:%d rdir = (%lf, %lf, %lf) %d\n", pixel_index, i, r.direction().x(), r.direction().y(), r.direction().z(), currentrs.d);

		color cc = raycolor(r, vec3(0, 0, 0), scene, &light);
		float max_sample_intensity = 50;
		cc = vec3(thrust::min(max_sample_intensity, cc[0]), thrust::min(max_sample_intensity, cc[1]), thrust::min(max_sample_intensity, cc[2]));
		sumcolor += cc;
	}

	//debug(sumcolor);
	fb[pixel_index] = sumcolor;
}

__device__ void load_cornell_box(Hittable** objects, HittableList** list, Camera** camera)
{
	//objects
	auto green = new CookTorrance(color(.12, .45, .15), 0.5);
	auto red = new CookTorrance(color(.65, .05, .05), 0.5);
	auto white = new CookTorrance(color(.73, .73, .73), 0.5);
	auto light = new DiffuseLight(color(15, 15, 15));
	auto gold = new CookTorrance(color(1, 0.71, 0.29), 0.05, vec3(1, 0.71, 0.29));
	auto sliver = new CookTorrance(color(0.91, 0.92, 0.92), 0.1, vec3(0.91, 0.92, 0.92));
	auto smoothsliver = new CookTorrance(color(0.91, 0.92, 0.92), 0.005, vec3(0.91, 0.92, 0.92));;

	*objects = new RectYZ(0, 555, 0, 555, 555, green);
	*(objects + 1) = new RectYZ(0, 555, 0, 555, 0, red);
	*(objects + 2) = new RectXZ(213, 343, 227, 332, 554, light);
	*(objects + 3) = new RectXZ(0, 555, 0, 555, 0, white);
	*(objects + 4) = new RectXZ(0, 555, 0, 555, 555, white);
	*(objects + 5) = new RectXY(0, 555, 0, 555, 555, white);

	Hittable* box1 = new Box(point3(0, 0, 0), point3(165, 330, 165), smoothsliver);
	box1 = new RotateY(box1, 15);
	box1 = new Translate(box1, vec3(265, 0, 295));
	*(objects + 6) = box1;

	Hittable* box2 = new Box(point3(0, 0, 0), point3(165, 165, 165), white);
	box2 = new RotateY(box2, -18);
	box2 = new Translate(box2, vec3(130, 0, 65));
	*(objects + 7) = box2;

	Hittable* sphere1 = new Sphere(point3(0, 0, 0), 75, sliver);
	sphere1 = new Translate(sphere1, vec3(250, 275, 245));
	*(objects + 8) = sphere1;

	//list
	*list = new HittableList(objects, 9);/////////////9

	//Camera
	color background = color(0, 0, 0);
	vec3 lookfrom = point3(278, 278, -800);
	vec3 lookat = vec3(278, 278, 0);
	float fov = 40.0;
	vec3 vup(0, 1, 0);
	auto dist_to_focus = 10.0;
	const auto aspect_ratio = 1.0;
	*camera = new Camera(lookfrom, lookat, vup, fov, aspect_ratio, 0, dist_to_focus);
}

__global__ void load_scene(Hittable** objects, HittableList** list, Camera** camera)
{
	if (blockIdx.x == 0 && threadIdx.x == 0)
	{
		load_cornell_box(objects, list, camera);
	}
}

__global__ void free_world(Hittable** objects, HittableList** list, Camera** camera)
{
	if (blockIdx.x == 0 && threadIdx.x == 0)
	{
		for (int i = 0; i < 9; i++)///////9
		{
			delete *(objects + i);
		}

		delete *list;

		delete *camera;
	}
}

void CudaRender(vec3* fb, int width, int height, int sampleTimes)
{
	Hittable** cudaList;
	HittableList** cudaObjects;
	Camera** cudaCamera;
	hiprandState* cudaRandState;

	// Load constant
	float inf = std::numeric_limits<float>::infinity();
	hipMemcpyToSymbol(HIP_SYMBOL(INF), &inf, sizeof(float));
	float pi = acos(-1);
	hipMemcpyToSymbol(HIP_SYMBOL(PI), &pi, sizeof(float));

	// Load scene
	CheckCudaError(hipMalloc((void**)&cudaList, 9 * sizeof(Hittable*)));//////9*
	CheckCudaError(hipMalloc((void**)&cudaObjects, sizeof(HittableList*)));
	CheckCudaError(hipMalloc((void**)&cudaCamera, sizeof(Camera*)));

	load_scene<<<1,1>>>(cudaList, cudaObjects, cudaCamera);
	CheckCudaError(hipDeviceSynchronize());
	CheckCudaError(hipGetLastError());

	//Load rand state
	CheckCudaError(hipMalloc((void**)&cudaRandState, width * height * sizeof(hiprandState)));
	
	//Rander
	int blockx = 16;
	int blocky = 16;

	dim3 block(blockx, blocky);
	dim3 grid(width / blockx + 1, height / blocky + 1);

	render << <grid, block >> > (fb, cudaRandState, cudaObjects, cudaCamera, width, height, sampleTimes);
	CheckCudaError(hipDeviceSynchronize());
	CheckCudaError(hipGetLastError());
	
	//Free
	free_world << <1, 1 >> > (cudaList, cudaObjects, cudaCamera);
	CheckCudaError(hipDeviceSynchronize());
	CheckCudaError(hipGetLastError());
}
