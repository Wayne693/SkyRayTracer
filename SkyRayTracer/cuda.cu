#include "hip/hip_runtime.h"
#include "cuda.cuh"
#include "utility.h"
#include "aarect.h"
#include "material.h"
#include "Box.h"
#include "Sphere.h"
#include "hittable.h"
#include "Camera.h"
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include "thrust/extrema.h"

__constant__ float INF;
__constant__ float PI;

__device__ color raycolor(Ray& ray, const color& backGround, const HittableList** scene, Hittable* light, int threadId)
{
	//color finalcolor = backGround;
	color n = vec3(0, 0, 0);
	color m = vec3(1, 1, 1);

	Ray r = ray;

	
	for (int i = 0; i < 100; i++)
	{
		HitRecord rec;
		rec.threadId = threadId;

		scatter_record srec;
		color curcolor;
		//debug(r.origin());
		//printf("**front** recaddress = %f id = %d\n", rec.u, threadId);
		//rec.u = 0;
		//change(rec, threadId);
		//printf("__ff = %d  %f\n", threadId, rec.u);

		if (!(*scene)->hit(r, 0.01f, INF, rec))
		{
			return n + m * backGround;
		}
		//printf("rec_address = %p ff = %d matptr = %p u = %lf v = %lf threadId = %d\n", &rec, rec.front_face, rec.mat_ptr, rec.u, rec.v, threadId);
		//printf("***p = (%lf %lf %lf) t = %lf ff = %d\n", rec.p.x(), rec.p.y(), rec.p.z(), rec.t, rec.front_face);

		
		color emitted = rec.mat_ptr->emitted(r, rec, rec.u, rec.v, rec.p);
		
		if (!rec.mat_ptr->scatter(r, rec, srec))
		{
			return n + m * emitted;
		}

		
		//if (srec.is_dielectric)
		//{
		//	//n = n + m * emitted;
		//	//m = m ;
		//	r = srec.speculer_ray;
		//	continue;
		//}

		//pdf* light_pdf = &HittablePDF(light, rec.p);
		pdf* mtl_pdf = srec.pdf_ptr;
		//MixPDF p(light_pdf, mtl_pdf);
		
		
		auto scattered = Ray(rec.p, mtl_pdf->generate(srec.curonb, r.randstate()), r.randstate());

		pdfrecord pdfrec;
		pdfrec.roughness = srec.roughness;
		pdfrec.n = rec.normal;
		//pdfrec.h = scattered.direction();//��������micronormal

		auto pdfval = mtl_pdf->value(srec.curonb, scattered.direction(), pdfrec);
		//auto ndotwi = dot(rec.normal.normalized(), scattered.direction().normalized());
		

		auto mtlval = rec.mat_ptr->scattering_pdf(r, rec, scattered);

		if (srec.is_dielectric)
		{
			//n = n + m * emitted;
			//m = m ;
			r = scattered;
			continue;
		}

		n = n + m * emitted;
		m = m * mtlval/ pdfval;//todo �����ndotwi��Ҫ��scattering_pdf��
		r = scattered;
	}

	return color(0, 0, 0);
}

__global__ void render(vec3* fb, HittableList** scene, Camera** camera, int width, int height, int spp)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
	{
		return;
	}

	int pixel_index = y * width + x;
	

	hiprandState currentrs;
	

	hiprand_init(20010418, pixel_index, 0, &currentrs);
	
	auto lm = new DiffuseLight(color(15, 15, 15));
	auto light = RectXZ(213, 343, 227, 332, 554, lm);
	
	color sumcolor = vec3(0, 0, 0);
	for (int i = 0; i < spp; i++)
	{
		float offset_x = Random(currentrs);
		float offset_y = Random(currentrs);
		//printf("%d %lf %lf\n", pixel_index, offset_x, offset_y);
		float u = float(x + offset_x) / float(width);
		float v = float(y + offset_y) / float(height);
		Ray r = (*camera)->GetRay(u, v, currentrs);
		//if (pixel_index == 10584)
		//	printf("idx = %d:%d rdir = (%lf, %lf, %lf) %d\n", pixel_index, i, r.direction().x(), r.direction().y(), r.direction().z(), currentrs.d);
		auto background = color(0.5, 0.7, 1.0);
		//auto background = color(0.8f, 0.8f, 0.8f);
		//auto background = color(0.f, 0.f, 0.f);
		color cc = raycolor(r, background, scene, &light, pixel_index);
		float max_sample_intensity = 50;
		cc = vec3(thrust::min(max_sample_intensity, cc[0]), thrust::min(max_sample_intensity, cc[1]), thrust::min(max_sample_intensity, cc[2]));
		sumcolor += cc;
	}

	//debug(sumcolor);
	fb[pixel_index] = sumcolor;
}

__device__ void load_cornell_box(Hittable** objects, HittableList** list, Camera** camera)
{
	//objects
	auto green = new BRDF(color(.12, .45, .15), 0.5);
	auto red = new BRDF(color(.65, .05, .05), 0.5);
	auto white = new BRDF(color(.73, .73, .73), 0.5);
	auto light = new DiffuseLight(color(15, 15, 15));
	auto gold = new BRDF(color(1, 0.71, 0.29), 0.05, vec3(1, 0.71, 0.29));
	auto sliver = new BRDF(color(0.91, 0.92, 0.92), 0.1, vec3(0.91, 0.92, 0.92));
	auto smoothsliver = new BRDF(color(0.91, 0.92, 0.92), 0.005, vec3(0.91, 0.92, 0.92));
	auto glass = new BTDF(color(0.8, 0.8, 0.8), 0.002, 1.3f);
	auto glass1 = new Dielectric(1.4f);

	*objects = new RectYZ(0, 555, 0, 555, 555, green);
	*(objects + 1) = new RectYZ(0, 555, 0, 555, 0, red);
	*(objects + 2) = new RectXZ(213, 343, 227, 332, 554, light);
	*(objects + 3) = new RectXZ(0, 555, 0, 555, 0, white);
	*(objects + 4) = new RectXZ(0, 555, 0, 555, 555, white);
	*(objects + 5) = new RectXY(0, 555, 0, 555, 555, white);

	Hittable* box1 = new Box(point3(0, 0, 0), point3(165, 330, 165), gold);
	box1 = new RotateY(box1, 15);
	box1 = new Translate(box1, vec3(265, 0, 295));
	*(objects + 6) = box1;

	Hittable* box2 = new Box(point3(0, 0, 0), point3(165, 165, 165), white);
	box2 = new RotateY(box2, -18);
	box2 = new Translate(box2, vec3(130, 0, 65));
	*(objects + 7) = box2;

	Hittable* sphere1 = new Sphere(point3(250, 275, 245), 75, glass);
	//sphere1 = new Translate(sphere1, vec3(250, 275, 245));
	*(objects + 8) = sphere1;

	//Hittable* sphere2 = new Sphere(point3(250, 275, 245), 30, sliver);
	//*(objects + 9) = sphere2;

	/*Hittable* box3 = new Box(point3(0, 0, 0), point3(75, 75, 75), glass1);
	box3 = new Translate(box3, vec3(250, 275, 245));
	*(objects) = box3;*/

	//list
	*list = new HittableList(objects, 9);/////////////9

	//Camera
	color background = color(0, 0, 0);
	vec3 lookfrom = point3(278, 278, -800);
	vec3 lookat = vec3(278, 278, 0);
	float fov = 40.0;
	vec3 vup(0, 1, 0);
	auto dist_to_focus = 10.0;
	const auto aspect_ratio = 1.0;
	*camera = new Camera(lookfrom, lookat, vup, fov, aspect_ratio, 0, dist_to_focus);
}

__global__ void load_scene(Hittable** objects, HittableList** list, Camera** camera)
{
	if (blockIdx.x == 0 && threadIdx.x == 0)
	{
		load_cornell_box(objects, list, camera);
	}
}

__global__ void free_world(Hittable** objects, HittableList** list, Camera** camera)
{
	if (blockIdx.x == 0 && threadIdx.x == 0)
	{
		for (int i = 0; i < 9; i++)///////9
		{
			delete *(objects + i);
		}

		delete *list;

		delete *camera;
	}
}

void CudaRender(vec3* fb, int width, int height, int sampleTimes)
{
	Hittable** cudaList;
	HittableList** cudaObjects;
	Camera** cudaCamera;
	//hiprandState* cudaRandState;

	// Load constant
	float inf = std::numeric_limits<float>::infinity();
	hipMemcpyToSymbol(HIP_SYMBOL(INF), &inf, sizeof(float));
	float pi = acos(-1);
	hipMemcpyToSymbol(HIP_SYMBOL(PI), &pi, sizeof(float));

	// Load scene
	CheckCudaError(hipMalloc((void**)&cudaList, 9 * sizeof(Hittable*)));//////9*
	CheckCudaError(hipMalloc((void**)&cudaObjects, sizeof(HittableList*)));
	CheckCudaError(hipMalloc((void**)&cudaCamera, sizeof(Camera*)));

	load_scene<<<1,1>>>(cudaList, cudaObjects, cudaCamera);
	CheckCudaError(hipDeviceSynchronize());
	CheckCudaError(hipGetLastError());

	//Load rand state
	//CheckCudaError(hipMalloc((void**)&cudaRandState, width * height * sizeof(hiprandState)));
	
	//Rander
	int blockx = 16;
	int blocky = 16;

	dim3 block(blockx, blocky);
	dim3 grid(width / blockx + 1, height / blocky + 1);

	render << <grid, block >> > (fb, cudaObjects, cudaCamera, width, height, sampleTimes);
	CheckCudaError(hipDeviceSynchronize());
	CheckCudaError(hipGetLastError());
	
	//Free
	free_world << <1, 1 >> > (cudaList, cudaObjects, cudaCamera);
	CheckCudaError(hipDeviceSynchronize());
	CheckCudaError(hipGetLastError());
}
